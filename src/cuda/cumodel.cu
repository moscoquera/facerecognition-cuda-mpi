#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <stdint.h>
#include <float.h>
#include "cuUtils.hpp"
#include "cuRoutines.hpp"
#include "cumodel.hpp"
#include "hipblas.h"
#include "hipsolver.h"


using namespace std;

const int MAX_STREAMS = 3;

double *alpha,*beta;

namespace cu{



EigenfacesModel::EigenfacesModel(int d,void (*dist_metric)(hipStream_t stream,double *, double *,double*, int)):
								BaseModel(d,dist_metric){
								hipblasCreate(&this->blasHandler);
								hipsolverDnCreate(&this->cusolverHandler);
								hipblasSetPointerMode(this->blasHandler,HIPBLAS_POINTER_MODE_DEVICE);
								double a=1,b=0;
								hipMalloc((void**)&alpha,sizeof(double));
								hipMalloc((void**)&beta,sizeof(double));
								hipMemcpy(alpha,&a,sizeof(double),hipMemcpyHostToDevice);
								hipMemcpy(beta,&b,sizeof(double),hipMemcpyHostToDevice);

	}
EigenfacesModel::~EigenfacesModel(){
	hipblasDestroy(this->blasHandler);
	hipsolverDnDestroy(this->cusolverHandler);
	hipDeviceReset();
}


void EigenfacesModel::compute(double* X, int* y, int n){
		double *d_V;
		double* cuX,*cuXC;
		int *cuY;

		this->streams=(hipStream_t*)malloc(sizeof(hipStream_t)*MAX_STREAMS);
		for(int xi=0;xi<MAX_STREAMS;xi++){
			hipStreamCreate(this->streams+xi);
		}

		hipMalloc((void**)&cuX,sizeof(double)*n*d);
		hipMalloc((void**)&cuY,sizeof(int)*n);
		hipMalloc((void**)&cuXC,sizeof(double)*n*d);
		hipMemcpyAsync(cuX,X,sizeof(double)*n*d,hipMemcpyHostToDevice,*this->streams);
		hipMemcpyAsync(cuY,y,sizeof(int)*n,hipMemcpyHostToDevice,*this->streams);


		//cambio cuX a column-major, m y n las invierto porque cuX originalmente es como si fuese la transpuesta
		CUBLAS_CHECK_RETURN(hipblasSetStream(this->blasHandler,*this->streams));
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,n,d,alpha,cuX,d,beta,cuXC,n,cuXC,n));



		pca(*this->streams,this->blasHandler,this->cusolverHandler,cuXC,this->num_components,n,this->d,this->W,d_V,this->mu);
		//this->y = cuY;
		this->y = y;


		this->projections=(double**)malloc(sizeof(double*)*n);
		this->nprojections=n;
		double *projections;
		hipMalloc((void**)&projections,sizeof(double)*n*this->num_components);

		//cuXC ya es cuXC-mu
		hipblasSetStream(this->blasHandler,*this->streams);
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,d,n,alpha,cuXC,n,beta,cuX,d,cuX,d));

		for(int xi=0;xi<n;xi++){
			double * proj=projections+(this->num_components*xi);
			*(this->projections+xi)=proj;
		}
		int step=ceil(n*1.0/MAX_STREAMS);
		for (int t=0;t<MAX_STREAMS;t++){
			hipblasSetStream(this->blasHandler,*(this->streams+t));
			hipblasDgemmStridedBatched(this->blasHandler,HIPBLAS_OP_N,HIPBLAS_OP_N,1,this->num_components,this->d,alpha,cuX+(d*step*t),1,this->d,this->W,this->d,0,beta,projections+(this->num_components*step*t),1,this->num_components,min(n-(step*t),step));
		}
		for (int t=0;t<MAX_STREAMS;t++){
			hipStreamSynchronize(*(this->streams+t));
		}
		hipFree(d_V);
		hipFree(cuX);
		hipFree(cuXC);

}


void EigenfacesModel::predict(double *X, int &faceClass, int &faceIdx){

	double* d_X;
	hipMalloc((void**)&d_X,sizeof(double)*this->d);
	hipMemcpy(d_X,X,sizeof(double)*this->d,hipMemcpyHostToDevice);

	double *d_Q;
	double* d_mins=0,*mins;
	hipMalloc((void**)&d_mins,sizeof(double)*this->nprojections);
	mins=(double*)malloc(sizeof(double)*this->nprojections);
	hipMalloc((void**)&d_Q,sizeof(double)*this->num_components*1);
	project(this->blasHandler,0,this->W,this->d,this->num_components,d_X,1,this->mu,d_Q);

	for(int i=0;i<this->nprojections;i++){
		double *pi = *(this->projections+i);
		this->dist_metric(0,pi,d_Q,d_mins+i,this->num_components);

	}
	hipDeviceSynchronize();
	hipMemcpy(mins,d_mins,sizeof(double)*this->nprojections,hipMemcpyDeviceToHost);
	double minDist= DBL_MAX;
	int minIdx=-1;

	for(int i=0;i<this->nprojections;i++){
		if (*(mins+i)<minDist){
			minDist=*(mins+i);
			minIdx=i;
		}
	}

	faceClass=(minIdx<0)?-1:this->y[minIdx];
	faceIdx=minIdx;
	hipFree(d_Q);
	hipFree(d_X);
	hipFree(d_mins);
}


double *EigenfacesModel::reconstructProjection(int data){

	double *d_R;
	reconstruct(this->blasHandler,this->W,this->d,this->num_components,*(this->projections+data),this->mu,d_R);
	double* R = (double*)malloc(sizeof(double)*this->d);
	hipMemcpy(R,d_R,sizeof(double)*this->d,hipMemcpyDeviceToHost);
	hipFree(d_R);
	return R;
}


FisherfacesModel::FisherfacesModel(int d,
							void (*dist_metric)(hipStream_t stream,double *, double *,double*, int)):
								BaseModel(d,dist_metric){
								hipblasCreate(&this->blasHandler);
								hipsolverDnCreate(&this->cusolverHandler);
								hipblasSetPointerMode(this->blasHandler,HIPBLAS_POINTER_MODE_DEVICE);
								double a=1,b=0;
								hipMalloc((void**)&alpha,sizeof(double));
								hipMalloc((void**)&beta,sizeof(double));
								hipMemcpy(alpha,&a,sizeof(double),hipMemcpyHostToDevice);
								hipMemcpy(beta,&b,sizeof(double),hipMemcpyHostToDevice);
	}

FisherfacesModel::~FisherfacesModel(){
		hipblasDestroy(this->blasHandler);
		hipsolverDnDestroy(this->cusolverHandler);
		hipDeviceReset();
}

void FisherfacesModel::compute(double* X, int* y, int n){
		double *d_V;
		double* cuX,*cuXC;

		this->streams=(hipStream_t*)malloc(sizeof(hipStream_t)*MAX_STREAMS);
		for(int xi=0;xi<MAX_STREAMS;xi++){
			hipStreamCreate(this->streams+xi);
		}

		hipMalloc((void**)&cuX,sizeof(double)*n*d);
		hipMalloc((void**)&cuXC,sizeof(double)*n*d);
		hipMemcpyAsync(cuX,X,sizeof(double)*n*d,hipMemcpyHostToDevice,*this->streams);
		//cambio cuX a column-major, m y n las invierto porque cuX originalmente es como si fuese la transpuesta
		hipblasSetStream(this->blasHandler,*this->streams);
		if (hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,n,d,alpha,cuX,d,beta,cuXC,n,cuXC,n)!=HIPBLAS_STATUS_SUCCESS){
			printf("holy shi!\n");
			exit(-1);
		}





		fisherfaces(this->streams,MAX_STREAMS,this->blasHandler,this->cusolverHandler, cuXC,y,this->num_components,n,this->d,this->W,d_V,this->mu,this->nfaces);


		this->y = y;



		this->projections=(double**)malloc(sizeof(double*)*n);
		this->nprojections=n;
		double *projections;
		hipMalloc((void**)&projections,sizeof(double)*n*this->nfaces);

		//cuXC ya es cuXC-mu
		hipblasSetStream(this->blasHandler,*streams);
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,d,n,alpha,cuXC,n,beta,cuX,d,cuX,d));

		for(int xi=0;xi<n;xi++){
			double * proj=projections+(this->nfaces*xi);
			*(this->projections+xi)=proj;
		}


		int step=ceil(n*1.0/MAX_STREAMS);
		for (int t=0;t<MAX_STREAMS;t++){
			hipblasSetStream(this->blasHandler,*(this->streams+t));
			hipblasDgemmStridedBatched(this->blasHandler,HIPBLAS_OP_N,HIPBLAS_OP_N,1,this->nfaces,this->d,alpha,cuX+(d*step*t),1,this->d,this->W,this->d,0,beta,projections+(this->nfaces*step*t),1,this->nfaces,min(n-(step*t),step));
		}
		for (int t=0;t<MAX_STREAMS;t++){
			hipStreamSynchronize(*(this->streams+t));
		}


		hipFree(d_V);
		hipFree(cuX);
		hipFree(cuXC);

}


void FisherfacesModel::predict(double *X, int &faceClass, int &faceIdx){

	double* d_X;
	hipMalloc((void**)&d_X,sizeof(double)*this->d);
	hipMemcpyAsync(d_X,X,sizeof(double)*this->d,hipMemcpyHostToDevice,*this->streams);

	double *d_Q;
	double* d_mins=0,*mins;
	hipMalloc((void**)&d_mins,sizeof(double)*this->nprojections);
	mins=(double*)malloc(sizeof(double)*this->nprojections);
	hipMalloc((void**)&d_Q,sizeof(double)*this->nfaces*1);
	project(this->blasHandler,0,this->W,this->d,this->nfaces,d_X,1,this->mu,d_Q);
	for(int i=0;i<this->nprojections;i++){
		double *pi = *(this->projections+i);
		this->dist_metric(*(this->streams+(i%MAX_STREAMS)),pi,d_Q,d_mins+i,this->nfaces);

	}
	hipDeviceSynchronize();
	hipMemcpy(mins,d_mins,sizeof(double)*this->nprojections,hipMemcpyDeviceToHost);

	double minDist= DBL_MAX;
	int minIdx=-1;

	for(int i=0;i<this->nprojections;i++){
		if (*(mins+i)<minDist){
			minDist=*(mins+i);
			minIdx=i;
		}
	}

	faceClass=(minIdx<0)?-1:this->y[minIdx];
	faceIdx=minIdx;
	hipFree(d_Q);
	hipFree(d_X);
	hipFree(d_mins);

}

double *FisherfacesModel::reconstructProjection(int data){

	double *d_R;
	reconstruct(this->blasHandler,this->W,this->d,this->nfaces,*(this->projections+data),this->mu,d_R);
	double* R = (double*)malloc(sizeof(double)*this->d);
	hipMemcpy(R,d_R,sizeof(double)*this->d,hipMemcpyDeviceToHost);
	hipFree(d_R);
	return R;
}

}
