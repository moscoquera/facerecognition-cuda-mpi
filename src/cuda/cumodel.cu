#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <stdint.h>
#include <float.h>
#include "cuUtils.hpp"
#include "cuRoutines.hpp"
#include "cumodel.hpp"
#include "hipblas.h"
#include "hipsolver.h"


using namespace std;

const int MAX_STREAMS = 3;

double *alpha,*beta;

namespace cu{



EigenfacesModel::EigenfacesModel(int d,void (*dist_metric)(hipStream_t stream,double *, double *,double*, int)):
								BaseModel(d,dist_metric){
								hipblasCreate(&this->blasHandler);
								hipsolverDnCreate(&this->cusolverHandler);
								hipblasSetPointerMode(this->blasHandler,HIPBLAS_POINTER_MODE_DEVICE);
								double a=1,b=0;
								hipMalloc((void**)&alpha,sizeof(double));
								hipMalloc((void**)&beta,sizeof(double));
								hipMemcpy(alpha,&a,sizeof(double),hipMemcpyHostToDevice);
								hipMemcpy(beta,&b,sizeof(double),hipMemcpyHostToDevice);
								this->streams=(hipStream_t*)malloc(sizeof(hipStream_t)*MAX_STREAMS);
								for(int xi=0;xi<MAX_STREAMS;xi++){
									hipStreamCreate(this->streams+xi);
								}

	}
EigenfacesModel::~EigenfacesModel(){
	hipblasDestroy(this->blasHandler);
	hipsolverDnDestroy(this->cusolverHandler);
	hipDeviceReset();
}


void EigenfacesModel::compute(double* X, int* y, int n){
		double *d_V;
		double* cuX,*cuXC;
		int *cuY;

		hipMalloc((void**)&cuX,sizeof(double)*n*d);
		hipMalloc((void**)&cuY,sizeof(int)*n);
		hipMalloc((void**)&cuXC,sizeof(double)*n*d);
		hipMemcpyAsync(cuX,X,sizeof(double)*n*d,hipMemcpyHostToDevice,*this->streams);
		hipMemcpyAsync(cuY,y,sizeof(int)*n,hipMemcpyHostToDevice,*this->streams);


		//cambio cuX a column-major, m y n las invierto porque cuX originalmente es como si fuese la transpuesta
		CUBLAS_CHECK_RETURN(hipblasSetStream(this->blasHandler,*this->streams));
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,n,d,alpha,cuX,d,beta,cuXC,n,cuXC,n));



		pca(*this->streams,this->blasHandler,this->cusolverHandler,cuXC,this->num_components,n,this->d,this->W,d_V,this->mu);
		//this->y = cuY;
		this->y = y;



		this->nprojections=n;
		hipMalloc((void**)&this->projections,sizeof(double)*n*this->num_components);

		//cuXC ya es cuXC-mu
		hipblasSetStream(this->blasHandler,*this->streams);
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,d,n,alpha,cuXC,n,beta,cuX,d,cuX,d));

		int step=ceil(n*1.0/MAX_STREAMS);
		for (int t=0;t<MAX_STREAMS;t++){
			hipblasSetStream(this->blasHandler,*(this->streams+t));
			hipblasDgemmStridedBatched(this->blasHandler,HIPBLAS_OP_N,HIPBLAS_OP_N,1,this->num_components,this->d,alpha,cuX+(d*step*t),1,this->d,this->W,this->d,0,beta,projections+(this->num_components*step*t),1,this->num_components,min(n-(step*t),step));
		}
		for (int t=0;t<MAX_STREAMS;t++){
			hipStreamSynchronize(*(this->streams+t));
		}
		hipFree(d_V);
		hipFree(cuX);
		hipFree(cuXC);

}

void EigenfacesModel::save(char* path){
	FILE *out = fopen(path, "wb");
	  if(out != NULL)
	  {
		  fwrite(&this->d,sizeof(int),1,out);
		  fwrite(&this->num_components,sizeof(int),1,out);
		  fwrite(&this->nprojections,sizeof(int),1,out);

			size_t to_go = this->nprojections;
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(this->y,sizeof(int),to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}

			to_go=this->d;
			double* hmu =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hmu,this->mu,sizeof(double)*to_go,hipMemcpyDeviceToHost);
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hmu,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hmu);


			to_go=this->d*this->num_components;
			double* hw =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hw,this->W,sizeof(double)*to_go,hipMemcpyDeviceToHost);

			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hw,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hw);

			to_go=this->nprojections*this->num_components;
			double* hpro =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hpro,this->projections,sizeof(double)*to_go,hipMemcpyDeviceToHost);
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hpro,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hpro);

		fclose(out);
	  }

}

void EigenfacesModel::load(char* path){
	FILE *in = fopen(path, "rb");
	  if(in != NULL)
	  {
		  fread(&this->d,sizeof(int),1,in);
		  fread(&this->num_components,sizeof(int),1,in);
		  fread(&this->nprojections,sizeof(int),1,in);
		  this->y=(int*)malloc(sizeof(int)*this->nprojections);
		  double *dmu = (double*)malloc(sizeof(double)*this->d);
		  double *dW = (double*)malloc(sizeof(double)*this->d*this->num_components);
		  double *dpro = (double*)malloc(sizeof(double)*this->nprojections*this->num_components);
		  CUDA_CHECK_RETURN(hipMalloc((void**)&this->mu,sizeof(double)*this->d));
		  CUDA_CHECK_RETURN(hipMalloc((void**)&this->W,sizeof(double)*this->d*this->num_components));
		  CUDA_CHECK_RETURN(hipMalloc((void**)&this->projections,sizeof(double)*this->nprojections*this->num_components));

		  fread(this->y,sizeof(int),this->nprojections,in);
		  fread(dmu,sizeof(double), this->d, in);
		  fread(dW,sizeof(double), this->d*this->num_components, in);
		  fread(dpro,sizeof(double), this->nprojections*this->num_components, in);

		  CUDA_CHECK_RETURN(hipMemcpy(this->mu,dmu,sizeof(double)*this->d,hipMemcpyHostToDevice));
		  CUDA_CHECK_RETURN(hipMemcpy(this->W,dW,sizeof(double)*this->d*this->num_components,hipMemcpyHostToDevice));
		  CUDA_CHECK_RETURN(hipMemcpy(this->projections,dpro,sizeof(double)*this->nprojections*this->num_components,hipMemcpyHostToDevice));

		  free(dW);
		  free(dmu);
		  free(dpro);

		  fclose(in);
	  }else{
		  printf("Load Error\n");
		  exit(-1);
	  }
}



double EigenfacesModel::predict(double *X, int &faceClass, int &faceIdx){

	double* d_X;
	hipMalloc((void**)&d_X,sizeof(double)*this->d);
	hipMemcpy(d_X,X,sizeof(double)*this->d,hipMemcpyHostToDevice);

	double *d_Q;
	double* d_mins=0,*mins;
	hipMalloc((void**)&d_mins,sizeof(double)*this->nprojections);
	mins=(double*)malloc(sizeof(double)*this->nprojections);
	hipMalloc((void**)&d_Q,sizeof(double)*this->num_components*1);
	project(this->blasHandler,0,this->W,this->d,this->num_components,d_X,1,this->mu,d_Q);

	for(int i=0;i<this->nprojections;i++){
		double *pi = this->projections+(i*this->num_components);
		this->dist_metric(0,pi,d_Q,d_mins+i,this->num_components);

	}
	hipDeviceSynchronize();
	hipMemcpy(mins,d_mins,sizeof(double)*this->nprojections,hipMemcpyDeviceToHost);
	double minDist= DBL_MAX;
	int minIdx=-1;

	for(int i=0;i<this->nprojections;i++){
		if (*(mins+i)<minDist){
			minDist=*(mins+i);
			minIdx=i;
		}
	}

	faceClass=(minIdx<0)?-1:this->y[minIdx];
	faceIdx=minIdx;
	hipFree(d_Q);
	hipFree(d_X);
	hipFree(d_mins);
	return minDist;
}


double *EigenfacesModel::reconstructProjection(int data){

	double *d_R;
	reconstruct(this->blasHandler,this->W,this->d,this->num_components,this->projections+(data*this->num_components),this->mu,d_R);
	double* R = (double*)malloc(sizeof(double)*this->d);
	hipMemcpy(R,d_R,sizeof(double)*this->d,hipMemcpyDeviceToHost);
	hipFree(d_R);
	return R;
}


FisherfacesModel::FisherfacesModel(int d,
							void (*dist_metric)(hipStream_t stream,double *, double *,double*, int)):
								BaseModel(d,dist_metric){
								hipblasCreate(&this->blasHandler);
								hipsolverDnCreate(&this->cusolverHandler);
								hipblasSetPointerMode(this->blasHandler,HIPBLAS_POINTER_MODE_DEVICE);
								double a=1,b=0;
								hipMalloc((void**)&alpha,sizeof(double));
								hipMalloc((void**)&beta,sizeof(double));
								hipMemcpy(alpha,&a,sizeof(double),hipMemcpyHostToDevice);
								hipMemcpy(beta,&b,sizeof(double),hipMemcpyHostToDevice);

								this->streams=(hipStream_t*)malloc(sizeof(hipStream_t)*MAX_STREAMS);
								for(int xi=0;xi<MAX_STREAMS;xi++){
									hipStreamCreate(this->streams+xi);
								}
	}

FisherfacesModel::~FisherfacesModel(){
		hipblasDestroy(this->blasHandler);
		hipsolverDnDestroy(this->cusolverHandler);
		hipDeviceReset();
}

void FisherfacesModel::compute(double* X, int* y, int n){
		double *d_V;
		double* cuX,*cuXC;


		hipMalloc((void**)&cuX,sizeof(double)*n*d);
		hipMalloc((void**)&cuXC,sizeof(double)*n*d);
		hipMemcpyAsync(cuX,X,sizeof(double)*n*d,hipMemcpyHostToDevice,*this->streams);
		//cambio cuX a column-major, m y n las invierto porque cuX originalmente es como si fuese la transpuesta
		hipblasSetStream(this->blasHandler,*this->streams);
		if (hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,n,d,alpha,cuX,d,beta,cuXC,n,cuXC,n)!=HIPBLAS_STATUS_SUCCESS){
			printf("holy shi!\n");
			exit(-1);
		}





		fisherfaces(this->streams,MAX_STREAMS,this->blasHandler,this->cusolverHandler, cuXC,y,this->num_components,n,this->d,this->W,d_V,this->mu,this->nfaces);


		this->y = y;




		this->nprojections=n;
		hipMalloc((void**)&this->projections,sizeof(double)*n*this->nfaces);

		//cuXC ya es cuXC-mu
		hipblasSetStream(this->blasHandler,*streams);
		CUBLAS_CHECK_RETURN(hipblasDgeam(this->blasHandler,HIPBLAS_OP_T,HIPBLAS_OP_N,d,n,alpha,cuXC,n,beta,cuX,d,cuX,d));


		int step=ceil(n*1.0/MAX_STREAMS);
		for (int t=0;t<MAX_STREAMS;t++){
			hipblasSetStream(this->blasHandler,*(this->streams+t));
			hipblasDgemmStridedBatched(this->blasHandler,HIPBLAS_OP_N,HIPBLAS_OP_N,1,this->nfaces,this->d,alpha,cuX+(d*step*t),1,this->d,this->W,this->d,0,beta,projections+(this->nfaces*step*t),1,this->nfaces,min(n-(step*t),step));
		}
		for (int t=0;t<MAX_STREAMS;t++){
			hipStreamSynchronize(*(this->streams+t));
		}


		hipFree(d_V);
		hipFree(cuX);
		hipFree(cuXC);

}


double FisherfacesModel::predict(double *X, int &faceClass, int &faceIdx){

	double* d_X;
	hipMalloc((void**)&d_X,sizeof(double)*this->d);
	hipMemcpyAsync(d_X,X,sizeof(double)*this->d,hipMemcpyHostToDevice,*this->streams);

	double *d_Q;
	double* d_mins=0,*mins;
	hipMalloc((void**)&d_mins,sizeof(double)*this->nprojections);
	mins=(double*)malloc(sizeof(double)*this->nprojections);
	hipMalloc((void**)&d_Q,sizeof(double)*this->nfaces*1);
	project(this->blasHandler,0,this->W,this->d,this->nfaces,d_X,1,this->mu,d_Q);
	for(int i=0;i<this->nprojections;i++){
		double *pi = this->projections+(this->nfaces*i);
		this->dist_metric(*(this->streams+(i%MAX_STREAMS)),pi,d_Q,d_mins+i,this->nfaces);

	}
	hipDeviceSynchronize();
	hipMemcpy(mins,d_mins,sizeof(double)*this->nprojections,hipMemcpyDeviceToHost);

	double minDist= DBL_MAX;
	int minIdx=-1;

	for(int i=0;i<this->nprojections;i++){
		if (*(mins+i)<minDist){
			minDist=*(mins+i);
			minIdx=i;
		}
	}

	faceClass=(minIdx<0)?-1:this->y[minIdx];
	faceIdx=minIdx;
	hipFree(d_Q);
	hipFree(d_X);
	hipFree(d_mins);
	return minDist;

}

double *FisherfacesModel::reconstructProjection(int data){

	double *d_R;
	reconstruct(this->blasHandler,this->W,this->d,this->nfaces,this->projections+(data*this->nfaces),this->mu,d_R);
	double* R = (double*)malloc(sizeof(double)*this->d);
	hipMemcpy(R,d_R,sizeof(double)*this->d,hipMemcpyDeviceToHost);
	hipFree(d_R);
	return R;
}

void FisherfacesModel::save(char* path){
	FILE *out = fopen(path, "wb");
	  if(out != NULL)
	  {
		  fwrite(&this->d,sizeof(int),1,out);
		  fwrite(&this->num_components,sizeof(int),1,out);
		  fwrite(&this->nprojections,sizeof(int),1,out);
		  fwrite(&this->nfaces,sizeof(int),1,out);

			size_t to_go = this->nprojections;
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(this->y,sizeof(int),to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}

			to_go=this->d;
			double* hmu =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hmu,this->mu,sizeof(double)*to_go,hipMemcpyDeviceToHost);
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hmu,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hmu);


			to_go=this->d*this->nfaces;
			double* hw =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hw,this->W,sizeof(double)*to_go,hipMemcpyDeviceToHost);

			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hw,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hw);

			to_go=this->nprojections*this->nfaces;
			double* hpro =(double*)malloc(sizeof(double)*to_go);
			hipMemcpy(hpro,this->projections,sizeof(double)*to_go,hipMemcpyDeviceToHost);
			while(to_go > 0)
			{
			  const size_t wrote = fwrite(hpro,sizeof(double), to_go, out);
			  if(wrote == 0)
				break;
			  to_go -= wrote;
			}
			free(hpro);

		fclose(out);
	  }

}

void FisherfacesModel::load(char* path){
	FILE *in = fopen(path, "rb");
	  if(in != NULL)
	  {
		  fread(&this->d,sizeof(int),1,in);
		  fread(&this->num_components,sizeof(int),1,in);
		  fread(&this->nprojections,sizeof(int),1,in);
		  fread(&this->nfaces,sizeof(int),1,in);
		  this->y=(int*)malloc(sizeof(int)*this->nprojections);
		  double *dmu = (double*)malloc(sizeof(double)*this->d);
		  double *dW = (double*)malloc(sizeof(double)*this->d*this->nfaces);
		  double *dpro = (double*)malloc(sizeof(double)*this->nprojections*this->nfaces);
		  hipMalloc((void**)&this->mu,sizeof(double)*this->d);
		  hipMalloc((void**)&this->W,sizeof(double)*this->d*this->nfaces);
		  hipMalloc((void**)&this->projections,sizeof(double)*this->nprojections*this->nfaces);

		  fread(this->y,sizeof(int),this->nprojections,in);
		  fread(dmu,sizeof(double), this->d, in);
		  fread(dW,sizeof(double), this->d*this->nfaces, in);
		  fread(dpro,sizeof(double), this->nprojections*this->nfaces, in);

		  hipMemcpy(this->mu,dmu,sizeof(double)*this->d,hipMemcpyHostToDevice);
		  hipMemcpy(this->W,dW,sizeof(double)*this->d*this->nfaces,hipMemcpyHostToDevice);
		  hipMemcpy(this->projections,dpro,sizeof(double)*this->nprojections*this->nfaces,hipMemcpyHostToDevice);

		  free(dW);
		  free(dmu);
		  free(dpro);

		  fclose(in);
	  }else{
		  printf("Load Error\n");
		  exit(-1);
	  }
}

}
